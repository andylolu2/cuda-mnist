#include <cutlass/gemm/device/gemm_batched.h>
#include <cutlass/numeric_types.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/reference/device/tensor_foreach.h>
#include <cutlass/util/tensor_view_io.h>

#include <iostream>

#include "cutlass/coord.h"
#include "cutlass/cutlass.h"
#include "cutlass/tensor_view.h"
#include "cutlass/util/reference/device/tensor_foreach.h"
#include "lib/3d_layout.h"
#include "lib/operators/matmul_bias_relu.h"
#include "lib/operators/matmul_bias_relu_bwd.h"

using Tensor3D = cutlass::HostTensor<cutlass::half_t, cutlass::layout::BatchedRowMajor>;

namespace detail {
    template <
        typename Element,  ///< Element type
        typename Layout>   ///< Layout function
    struct TensorMulFunc {
        /// View type
        using TensorView = cutlass::TensorView<Element, Layout>;

        /// Coordinate in tensor's index space
        using TensorCoord = typename TensorView::TensorCoord;

        /// Parameters structure
        struct Params {
            //
            // Data members
            //

            TensorView view;
            Element c;

            //
            // Methods
            //

            Params(TensorView view_ = TensorView(), Element c_ = Element(0)) : view(view_), c(c_) {}
        };

        //
        // Data members
        //
        Params params;

        //
        // Methods
        //
        CUTLASS_DEVICE
        TensorMulFunc(Params const &params) : params(params) {}

        CUTLASS_DEVICE
        void operator()(TensorCoord const &coord) {
            Element const &value = params.view.at(coord);
            params.view.at(coord) = params.c * value;
        }
    };
};  // namespace detail

template <typename Element, typename Layout>
void multiply(cutlass::TensorView<Element, Layout> view, Element c) {
    using Func = detail::TensorMulFunc<Element, Layout>;
    using Params = typename Func::Params;

    cutlass::reference::device::TensorForEach<Func, Layout::kRank, Params>(
        view.extent(), Params(view, c)
    );
};

hipError_t cutlass_strided_batched_sgemm(
    Tensor3D &A, Tensor3D &B, Tensor3D &C, Tensor3D &D, cutlass::half_t alpha, cutlass::half_t beta
) {
    using Gemm = cutlass::gemm::device::GemmBatched<
        cutlass::half_t, cutlass::layout::RowMajor, cutlass::half_t, cutlass::layout::RowMajor,
        cutlass::half_t, cutlass::layout::RowMajor>;
    Gemm gemm_op;

    auto batch_stride_A = A.layout().stride_batch();
    auto batch_stride_B = B.layout().stride_batch();
    auto batch_stride_C = C.layout().stride_batch();
    auto batch_stride_D = D.layout().stride_batch();
    auto n_batches = A.extent().at(0);
    auto M = A.extent().at(1);
    auto N = A.extent().at(2);
    auto K = B.extent().at(2);

    cutlass::Status status = gemm_op(
        {{M, N, K},
         //   A.device_ref(),
         {A.device_data(), A.layout().stride_row()},
         batch_stride_A,
         //   B.device_ref(),
         {B.device_data(), B.layout().stride_row()},
         batch_stride_B,
         //   C.device_ref(),
         {C.device_data(), C.layout().stride_row()},
         batch_stride_C,
         //   D.device_ref(),
         {D.device_data(), D.layout().stride_row()},
         batch_stride_D,
         {alpha, beta},
         n_batches}
    );

    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

int main() {
    int batch = 2;
    int D_in = 16;
    int D_out = 16;

    cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> x({batch, D_in});
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> w({D_out, D_in});
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::PackedVectorLayout> b(cutlass::Coord<1>{
        D_out});
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> y({batch, D_out});

    cutlass::reference::device::TensorFillRandomGaussian(x.device_view(), 42);
    // cutlass::reference::device::TensorFillRandomUniform(w.device_view(), 41, 1.0_hf, -1.0_hf);
    cutlass::reference::device::TensorFillIdentity(w.device_view());
    // cutlass::reference::device::TensorFill(z.device_view(), 1.5_hf);

    // multiply<cutlass::half_t, Tensor3D::Layout>(x.device_view(), -1.0_hf);

    x.sync_host();
    w.sync_host();
    y.sync_host();

    std::cout << "x:\n" << x.host_view() << "\n" << std::endl;
    std::cout << "w:\n" << w.host_view() << "\n" << std::endl;
    std::cout << "y:\n" << y.host_view() << "\n" << std::endl;

    lib::ops::batched_matmul_bias_relu(x, w, b, y);

    y.sync_host();

    std::cout << "y:\n" << y.host_view() << "\n" << std::endl;

    cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> dx({batch, D_in});
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> dw({D_out, D_in});
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::PackedVectorLayout> db(cutlass::Coord<1>{
        D_out});
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> dy({batch, D_out});
    cutlass::HostTensor<float, cutlass::layout::RowMajor> d_after_bias({batch, D_out});
    cutlass::HostTensor<float, cutlass::layout::PackedVectorLayout> b_(cutlass::Coord<1>{D_out});

    lib::ops::batched_matmul_bias_relu_bwd(x, w, b_, dy, db, dw, dx, d_after_bias);

    return 0;
}